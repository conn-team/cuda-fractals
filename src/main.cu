#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include "GL/glew.h"
#include "GL/freeglut.h"
#include "cuda_gl_interop.h"

#include "cuda_helper.hpp"

struct hipGraphicsResource *cudaViewBuffer;
GLuint viewBuffer, viewTexture;
int width, height;

__global__ void gradient(uint32_t *img, int width, int height) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int i = y*width + x;
        uint8_t val = uint8_t(255 * x / width);
        img[i] = 0xFF000000 | (val << 16) | ((255-val) << 8);
    }
}

void renderImage(uint32_t *mapped) {
    constexpr uint32_t blockSize = 32;
    uint32_t xBlocks = (width+blockSize-1) / blockSize;
    uint32_t yBlocks = (height+blockSize-1) / blockSize;
    gradient<<<{xBlocks, yBlocks}, {blockSize, blockSize}>>>(mapped, width, height);
    gpuErrchk(hipDeviceSynchronize());
}

void onRender() {
    void *mapped;
    size_t mappedSize;
    gpuErrchk(hipGraphicsMapResources(1, &cudaViewBuffer, 0));
    gpuErrchk(hipGraphicsResourceGetMappedPointer(&mapped, &mappedSize, cudaViewBuffer));
    renderImage(reinterpret_cast<uint32_t*>(mapped));
    gpuErrchk(hipGraphicsUnmapResources(1, &cudaViewBuffer, 0));

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, viewBuffer);
    glBindTexture(GL_TEXTURE_2D, viewTexture);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_BGRA, GL_UNSIGNED_BYTE, nullptr);

    glBegin(GL_QUADS);
        glTexCoord2f(0, 0); glVertex2f(0, 0);
        glTexCoord2f(1, 0); glVertex2f(1, 0);
        glTexCoord2f(1, 1); glVertex2f(1, 1);
        glTexCoord2f(0, 1); glVertex2f(0, 1);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    glFlush();
}

void onReshape(int w, int h) {
    width = w;
    height = h;

    // Free old buffers
    if (viewBuffer) {
        hipGraphicsUnregisterResource(cudaViewBuffer);
        glDeleteBuffers(1, &viewBuffer);
        viewBuffer = 0;
    }
    if (viewTexture) {
        glDeleteTextures(1, &viewTexture);
        viewTexture = 0;
    }

    // Allocate texture
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &viewTexture);
    glBindTexture(GL_TEXTURE_2D, viewTexture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0, GL_BGRA, GL_UNSIGNED_BYTE, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);

    // Allocate and register PBO
    glGenBuffers(1, &viewBuffer);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, viewBuffer);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, width*height*4, nullptr, GL_DYNAMIC_COPY);
    gpuErrchk(hipGraphicsGLRegisterBuffer(&cudaViewBuffer, viewBuffer, cudaGraphicsMapFlagsWriteDiscard));
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // Setup scene
    glViewport(0, 0, width, height);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glOrtho(0, 1, 0, 1, 0, 1);
}

int main(int argc, char **argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_SINGLE);
    glutInitWindowSize(500, 500);
    glutInitWindowPosition(100, 100);
    glutCreateWindow("cuda-fractals");

    glewInit();

    glutDisplayFunc(onRender);
    glutReshapeFunc(onReshape);
    glutMainLoop();
    return 0;
}
