#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <iostream>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include "cuda_helper.hpp"
#include "renderer.hpp"
#include "mandelbrot.hpp"
#include "julia.hpp"

constexpr double SUPERSAMPLING_RATIO = 1;
constexpr double ZOOM_SPEED = 1.2;

struct hipGraphicsResource *cudaViewBuffer;
GLuint viewBuffer, viewTexture;
int width, height;

int lastX, lastY;
bool isMoving = false;

Renderer<Mandelbrot> mandelbrotView;
Renderer<Julia> juliaView;
std::vector<BaseRenderer*> views = { &mandelbrotView, &juliaView };
int fractalIdx = 0;

BaseRenderer& getView() {
    return *views[fractalIdx];
}

void updateTitle() {
    std::ostringstream tmp;
    tmp << "cuda-fractals (zoom: " << (1 / getView().getScale()) << ", maxIters: " << getView().maxIters << ")";
    std::string title = tmp.str();
    glutSetWindowTitle(title.c_str());
}

void printCoordinates() {
    std::cout << std::fixed << std::setprecision(std::max(0L, lround(-log10(getView().getScale()))) + 5);
    std::cout << "center real: " << getView().center.x << std::endl;
    std::cout << "center imag: " << getView().center.y << std::endl;
    std::cout << std::scientific << std::setprecision(5);
    std::cout << "scale: " << getView().getScale() << std::endl << std::endl;
}

void onRender() {
    // Map PBO to CUDA
    void *devImage;
    size_t mappedSize;
    gpuErrchk(hipGraphicsMapResources(1, &cudaViewBuffer, 0));
    gpuErrchk(hipGraphicsResourceGetMappedPointer(&devImage, &mappedSize, cudaViewBuffer));

    // Render image
    getView().render(reinterpret_cast<Color*>(devImage));

    // Unmap PBO
    gpuErrchk(hipGraphicsUnmapResources(1, &cudaViewBuffer, 0));

    // Copy PBO to texture
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, viewBuffer);
    glBindTexture(GL_TEXTURE_2D, viewTexture);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, getView().width, getView().height, GL_BGRA, GL_UNSIGNED_BYTE, nullptr);

    // Render full-screen quad
    glBegin(GL_QUADS);
        glTexCoord2f(0, 0); glVertex2f(0, 0);
        glTexCoord2f(1, 0); glVertex2f(1, 0);
        glTexCoord2f(1, 1); glVertex2f(1, 1);
        glTexCoord2f(0, 1); glVertex2f(0, 1);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    glFlush();
}

void onMouse(int button, int state, int x, int y) {
    if (button == 3 || button == 4) {
        if (state == GLUT_UP) {
            return;
        }

        double zoom = (button == 3 ? ZOOM_SPEED : 1/ZOOM_SPEED);
        double dx = 2.0*x/width - 1;
        double dy = 2.0*y/width - double(height)/width;

        getView().center.x -= dx*(zoom-1)*getView().getScale();
        getView().center.y += dy*(zoom-1)*getView().getScale();
        getView().setScale(getView().getScale() * zoom);

        updateTitle();
        glutPostRedisplay();
    } else if (button == GLUT_LEFT_BUTTON) {
        isMoving = (state == GLUT_DOWN);
    } else if (button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN) {
        printCoordinates();
    }
}

void onMotion(int x, int y) {
    int dx = x-lastX, dy = y-lastY;
    lastX = x;
    lastY = y;

    if (isMoving) {
        getView().center.x -= 2*dx*getView().getScale()/width;
        getView().center.y += 2*dy*getView().getScale()/width;
        glutPostRedisplay();
    }
}

void onKeyboard(unsigned char key, int, int) {
    if (key == 's') {
        getView().useSeriesApproximation = !getView().useSeriesApproximation;
    } else if (key == 'i') {
        getView().useSmoothing = !getView().useSmoothing;
    } else if (key == 'r') {
        getView().center = {-0.7, 0};
        getView().setScale(1.5);
        getView().maxIters = 250;    
    } else {
        return;
    }
    
    updateTitle();
    glutPostRedisplay();
}

void onSpecialKeyboard(int key, int, int) {
    if (key == GLUT_KEY_UP) {
        getView().maxIters += 250;
    } else if (key == GLUT_KEY_DOWN) {
        getView().maxIters = std::max(getView().maxIters-250, 0);
    } else if (key == GLUT_KEY_RIGHT) {
        fractalIdx = (fractalIdx + 1) % views.size();
    } else if (key == GLUT_KEY_LEFT) {
        if (--fractalIdx < 0) {
            fractalIdx = views.size() - 1;
        }
    } else {
        return;
    }

    glutPostRedisplay();
    updateTitle();
}

void onReshape(int w, int h) {
    width = w;
    height = h;

    for (auto view : views) {
        view->width = int(lround(w*SUPERSAMPLING_RATIO));
        view->height = int(lround(h*SUPERSAMPLING_RATIO));
    }

    // Free old buffers
    if (viewBuffer) {
        hipGraphicsUnregisterResource(cudaViewBuffer);
        glDeleteBuffers(1, &viewBuffer);
        viewBuffer = 0;
    }
    if (viewTexture) {
        glDeleteTextures(1, &viewTexture);
        viewTexture = 0;
    }

    // Allocate texture
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &viewTexture);
    glBindTexture(GL_TEXTURE_2D, viewTexture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, getView().width, getView().height, 0, GL_BGRA, GL_UNSIGNED_BYTE, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);

    // Allocate and register PBO
    glGenBuffers(1, &viewBuffer);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, viewBuffer);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, getView().width*getView().height*4, nullptr, GL_DYNAMIC_COPY);
    gpuErrchk(hipGraphicsGLRegisterBuffer(&cudaViewBuffer, viewBuffer, cudaGraphicsMapFlagsWriteDiscard));
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // Setup scene
    glViewport(0, 0, width, height);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glOrtho(0, 1, 0, 1, 0, 1);
}

int main(int argc, char **argv) {
    for (auto view : views) {
        view->center = {-0.7, 0};
        view->setScale(1.5);
        view->maxIters = 250;
    }

    // NaN series breaking zoom
    // mandelbrotView.maxIters = 1250;
    // mandelbrotView.setScale(2.80969e-104);
    // mandelbrotView.center.x = BigFloat("-0.4968141896256946114192256490519277983341532366871239006397328938102282969608105818918291392319167436980711814");
    // mandelbrotView.center.y = BigFloat("-0.6359556404531552450576807825161928936851063796124890071820830821264561315128502700495367013919277125766152971");

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_SINGLE);
    glutInitWindowSize(800, 600);
    glutInitWindowPosition(200, 200);
    glutCreateWindow("cuda-fractals");

    glewInit();
    updateTitle();

    glutDisplayFunc(onRender);
    glutMouseFunc(onMouse);
    glutMotionFunc(onMotion);
    glutPassiveMotionFunc(onMotion);
    glutKeyboardFunc(onKeyboard);
    glutSpecialFunc(onSpecialKeyboard);
    glutReshapeFunc(onReshape);
    glutMainLoop();
    return 0;
}
