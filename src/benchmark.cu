#include <chrono>
#include <iostream>
#include <iomanip>

#include "cuda_helper.hpp"
#include "benchmark.hpp"
#include "renderer.hpp"
#include "mandelbrot.hpp"

constexpr int WIDTH = 800;
constexpr int HEIGHT = 600;
constexpr int REPEAT = 20;

using Clock = std::chrono::high_resolution_clock;
using Time = std::chrono::time_point<Clock>;

void benchmarkLocation(const char *name, const char *centerX, const char *centerY, double scale, int iters) {
    Renderer<Mandelbrot> view;
    view.width = WIDTH;
    view.height = HEIGHT;

    view.maxIters = iters;
    view.setScale(scale);
    view.center.x = BigFloat(centerX);
    view.center.y = BigFloat(centerY);

    CudaArray<Color> image(WIDTH*HEIGHT);

    Time start = Clock::now();

    for (int i = 0; i < REPEAT; i++) {
        view.render(image.data());
        gpuErrchk(hipDeviceSynchronize());
    }

    Time finish = Clock::now();
    int time = std::chrono::duration_cast<std::chrono::milliseconds>(finish - start).count();

    std::cout << std::setfill(' ') << std::setw(25) << name << "  ";
    std::cout << std::fixed << std::setprecision(3) << double(time)/1000 << "s" << std::endl;
}

void runBenchmarks() {
    benchmarkLocation("default", "-0.7", "0", 1.5, 250);
    benchmarkLocation("default-high-iters", "-0.7", "0", 1.5, 5000);

    benchmarkLocation(
        "deep",
        "-0.00608110996414361738609067953992572361899653523214435533970021571665715422246314237437352765062921732023885",
        "0.80710509130889151108972120096753442734230451562765569259191208053695855881108570059709735793308459433434174",
        6.23362e-103, 15000
    );

    benchmarkLocation(
        "mosaic",
        "0.372137738770323258373356630885867793129508737859268",
        "-0.090398245434178161692952411151009819302665482561413",
        3.47252e-47, 20000
    );

    benchmarkLocation(
        "hard",
        "-1.47994622332507888020258065344256383359082887482853327232891946750450142804155145810212315771521365103554594354207816734895388578734190261250998672",
        "0.00090139732902035398019779186619717356625156681804510241106763038648869228718904914562158443602693421876352757729063180945479661811076742458032279",
        6e-141, 45000
    );
}
